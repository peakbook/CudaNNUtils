#include "hip/hip_runtime.h"
__device__ void equiv(hipDoubleComplex *dst,const Quaternion *src, const int w, const int h);
__device__ void equiv(Quaternion *dst,const hipDoubleComplex *src, const int w, const int h);
__device__ void equiv(hipFloatComplex *dst,const Quaternionf *src, const int w, const int h);
__device__ void equiv(Quaternionf *dst,const hipFloatComplex *src, const int w, const int h);
__device__ void transpose_pre(hipFloatComplex *dst, const int w, const int h);
__device__ void transpose_pre(hipDoubleComplex *dst, const int w, const int h);
template<typename T> __device__ void transpose(T *dst, const T *src, const int w, const int h);
template<typename T> __device__ void ctranspose(T *dst, const T *src, const int w, const int h);

__device__ void equiv(hipDoubleComplex *dst, const Quaternion *src, const int w, const int h)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int idxc1 = j*2*w+i;
    int idxc2 = j*2*w+i+w;
    int idxc3 = (j+h)*2*w+i;
    int idxc4 = (j+h)*2*w+i+w;
    int idxq = j*w+i;

    if(i < w && j< h)
    {
        Quaternion val = src[idxq];
        dst[idxc1] = make_double2(val.x, val.y);
        dst[idxc2] = make_double2(-val.z, val.w);
        dst[idxc3] = make_double2(val.z, val.w);
        dst[idxc4] = make_double2(val.x, -val.y);
    }
}

__device__ void equiv(hipFloatComplex *dst, const Quaternionf *src, const int w, const int h)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int idxc1 = j*2*w+i;
    int idxc2 = j*2*w+i+w;
    int idxc3 = (j+h)*2*w+i;
    int idxc4 = (j+h)*2*w+i+w;
    int idxq = j*w+i;

    if(i < w && j< h)
    {
        Quaternionf val = src[idxq];
        dst[idxc1] = make_float2(val.x, val.y);
        dst[idxc2] = make_float2(-val.z, val.w);
        dst[idxc3] = make_float2(val.z, val.w);
        dst[idxc4] = make_float2(val.x, -val.y);
    }
}

__device__ void equiv(Quaternion *dst, const hipDoubleComplex *src, const int w, const int h)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int idxc1 = j*2*w+i;
    int idxc3 = (j+h)*2*w+i;
    int idxq = j*w+i;

    if(i < w && j< h)
    {
        hipDoubleComplex a = src[idxc1];
        hipDoubleComplex b = src[idxc3];
        dst[idxq] = quaternion(hipCreal(a), hipCimag(a), hipCreal(b), hipCimag(b));
    }
}

__device__ void equiv(Quaternionf *dst, const hipFloatComplex *src, const int w, const int h)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int idxc1 = j*2*w+i;
    int idxc3 = (j+h)*2*w+i;
    int idxq = j*w+i;

    if(i < w && j< h)
    {
        hipFloatComplex a = src[idxc1];
        hipFloatComplex b = src[idxc3];
        dst[idxq] = quaternion(hipCrealf(a), hipCimagf(a), hipCrealf(b), hipCimagf(b));
    }
}

__device__ void transpose_pre(hipDoubleComplex *dst, const int w, const int h)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int hw = w>>1;
    int hh = h>>1;
    int idx = j*w+i;

    if(i < w && j< h)
    {
        if (( i >= hw && j < hh) || ( i < hw && j >= hh)){
            hipDoubleComplex val = dst[idx];
            dst[idx] = make_double2(-val.x, val.y);
        }
    }
}

__device__ void transpose_pre(hipFloatComplex *dst, const int w, const int h)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int hw = w>>1;
    int hh = h>>1;
    int idx = j*w+i;

    if(i < w && j< h)
    {
        if (( i >= hw && j < hh) || ( i < hw && j >= hh)){
            hipFloatComplex val = dst[idx];
            dst[idx] = make_float2(-val.x, val.y);
        }
    }
}

template<typename T>
__device__ void transpose(T *dst, const T *src, const int w, const int h)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int idx_dst = j*w+i;
    int idx_src = i*h+j;

    if(i < w && j< h)
    {
        dst[idx_dst] = src[idx_src];
    }
}

template<typename T>
__device__ void ctranspose(T *dst, const T *src, const int w, const int h)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int idx_dst = j*w+i;
    int idx_src = i*h+j;

    if(i < w && j< h)
    {
        dst[idx_dst] = qconj(src[idx_src]);
    }
}

extern "C"{
    void __global__ q2c_float(hipFloatComplex *dst, const Quaternionf *src, const int w, const int h) { equiv(dst,src,w,h);}
    void __global__ q2c_double(hipDoubleComplex *dst, const Quaternion *src, const int w, const int h) { equiv(dst,src,w,h);}
    void __global__ c2q_float(Quaternionf *dst, const hipFloatComplex *src, const int w, const int h) { equiv(dst,src,w,h);}
    void __global__ c2q_double(Quaternion *dst, const hipDoubleComplex *src, const int w, const int h) { equiv(dst,src,w,h);}
    void __global__ transpose_pre_float(hipFloatComplex *dst, const int w, const int h) { transpose_pre(dst,w,h);}
    void __global__ transpose_pre_double(hipDoubleComplex *dst, const int w, const int h) { transpose_pre(dst,w,h);}
    void __global__ transpose_float(Quaternionf *dst, const Quaternionf *src, const int w, const int h) { transpose(dst,src,w,h);}
    void __global__ transpose_double(Quaternion *dst, const Quaternion *src, const int w, const int h) { transpose(dst,src,w,h);}
    void __global__ ctranspose_float(Quaternionf *dst, const Quaternionf *src, const int w, const int h) { ctranspose(dst,src,w,h);}
    void __global__ ctranspose_double(Quaternion *dst, const Quaternion *src, const int w, const int h) { ctranspose(dst,src,w,h);}
}
